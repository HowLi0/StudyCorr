#include "hip/hip_runtime.h"
#include "sc_sift_affine.h"
#include <cassert>
#include <cmath>
#include <vector>
#include <algorithm>
#include <nanoflann.hpp>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <nanoflann.hpp>

namespace StudyCorr_GPU {

// KDTree adaptor for nanoflann
struct SiftKeypointCloud {
    const SiftFeature2D* pts;
    size_t num;
    inline size_t kdtree_get_point_count() const { return num; }
    inline float kdtree_get_pt(const size_t idx, int dim) const {
        return (dim == 0) ? pts[idx].x : pts[idx].y;
    }
    template <class BBOX> bool kdtree_get_bbox(BBOX&) const { return false; }
};

// nanoflann KDTree: use explicit size_t for index type
typedef nanoflann::KDTreeSingleIndexAdaptor<
    nanoflann::L2_Simple_Adaptor<float, SiftKeypointCloud>,
    SiftKeypointCloud, 2, uint32_t> SiftKDTree;

// CUDA RANSAC仿射核，OpenCorr风格
struct SiftAffineRansacParam {
    int trial_number;
    int sample_number;
    float error_threshold;
    int min_inlier;
};

__device__ float norm2(float x, float y) { return sqrtf(x*x + y*y); }

__global__ void estimate_affine_opencorr_kernel(
    const SiftFeature2D* ref_kp,
    const SiftFeature2D* tar_kp,
    const int* poi_neighbor_idx, // [N*max_neighbor]
    const float* poi_neighbor_dist, // [N*max_neighbor]
    int max_neighbor,
    CudaPOI2D* pois,
    int N,
    SiftAffineRansacParam param,
    unsigned long long seed
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;
    CudaPOI2D& poi = pois[idx];

    // 1. 邻居数据
    int neighbor_num = 0;
    int neighbor_idx[30];
    float neighbor_dist[30];
    for (int i = 0; i < max_neighbor; ++i) {
        int nidx = poi_neighbor_idx[idx*max_neighbor + i];
        if (nidx < 0) break;
        neighbor_idx[i] = nidx;
        neighbor_dist[i] = poi_neighbor_dist[idx*max_neighbor + i];
        neighbor_num++;
    }

    if (neighbor_num < param.sample_number) {
        poi.result.zncc = -1.f;
        poi.result.feature = 0;
        return;
    }

    // 2. 局部坐标转换
    float ref_X[30], ref_Y[30], tar_X[30], tar_Y[30];
    for (int i = 0; i < neighbor_num; ++i) {
        int kp_idx = neighbor_idx[i];
        ref_X[i] = ref_kp[kp_idx].x - poi.x;
        ref_Y[i] = ref_kp[kp_idx].y - poi.y;
        tar_X[i] = tar_kp[kp_idx].x - poi.x;
        tar_Y[i] = tar_kp[kp_idx].y - poi.y;
    }

    // 3. RANSAC洗牌采样
    int best_inlier = 0, best_trial_counter = 0;
    float best_affine[6] = {0};
    int best_inlier_indices[30];
    float best_mean_error = 1e10f;

    hiprandState state;
    hiprand_init(seed + idx, 0, 0, &state);

    int candidate_idx[30];
    for (int i = 0; i < neighbor_num; ++i) candidate_idx[i] = i;

    int trial_counter = 0;
    do {
        // 洗牌采样
        for (int i = neighbor_num-1; i > 0; --i) {
            int j = hiprand(&state) % (i+1);
            int tmp = candidate_idx[i]; candidate_idx[i] = candidate_idx[j]; candidate_idx[j] = tmp;
        }
        // 构造最小二乘
        float A[9] = {0}, Bx[3] = {0}, By[3] = {0};
        for (int s = 0; s < param.sample_number; ++s) {
            int k = candidate_idx[s];
            float x = ref_X[k], y = ref_Y[k], tx = tar_X[k], ty = tar_Y[k];
            float v[3] = {x, y, 1.f};
            for (int r = 0; r < 3; ++r) for (int c = 0; c < 3; ++c)
                A[r*3+c] += v[r]*v[c];
            for (int r = 0; r < 3; ++r) {
                Bx[r] += v[r]*tx;
                By[r] += v[r]*ty;
            }
        }
        float invA[9];
        float det = A[0]*A[4]*A[8] + A[1]*A[5]*A[6] + A[2]*A[3]*A[7]
                  - A[0]*A[5]*A[7] - A[1]*A[3]*A[8] - A[2]*A[4]*A[6];
        bool ok = fabs(det) >= 1e-6f;
        if (!ok) { trial_counter++; continue; }
        invA[0]=(A[4]*A[8]-A[5]*A[7])/det; invA[1]=(A[2]*A[7]-A[1]*A[8])/det; invA[2]=(A[1]*A[5]-A[2]*A[4])/det;
        invA[3]=(A[5]*A[6]-A[3]*A[8])/det; invA[4]=(A[0]*A[8]-A[2]*A[6])/det; invA[5]=(A[2]*A[3]-A[0]*A[5])/det;
        invA[6]=(A[3]*A[7]-A[4]*A[6])/det; invA[7]=(A[1]*A[6]-A[0]*A[7])/det; invA[8]=(A[0]*A[4]-A[1]*A[3])/det;
        float affine_x[3]={0}, affine_y[3]={0};
        for (int r=0;r<3;++r) for(int j=0;j<3;++j) {
            affine_x[r]+=invA[r*3+j]*Bx[j]; affine_y[r]+=invA[r*3+j]*By[j];
        }
        // 一致集判据
        int inlier=0;
        int inlier_indices[30];
        float mean_error = 0.0f;
        for(int i=0;i<neighbor_num;++i) {
            float x=ref_X[i], y=ref_Y[i];
            float tx=affine_x[0]*x + affine_x[1]*y + affine_x[2];
            float ty=affine_y[0]*x + affine_y[1]*y + affine_y[2];
            float err=norm2(tx-tar_X[i],ty-tar_Y[i]);
            if(err<param.error_threshold) {
                inlier_indices[inlier] = i;
                mean_error += err;
                inlier++;
            }
        }
        mean_error = inlier > 0 ? mean_error/inlier : 1e10f;
        if (inlier > best_inlier ||
            (inlier == best_inlier && mean_error < best_mean_error))
        {
            best_inlier = inlier;
            best_mean_error = mean_error;
            for (int k = 0; k < inlier; ++k) best_inlier_indices[k] = inlier_indices[k];
            for (int i=0;i<3;++i) { best_affine[i]=affine_x[i]; best_affine[i+3]=affine_y[i]; }
            best_trial_counter = trial_counter;
        }
        trial_counter++;
    } while (trial_counter < param.trial_number &&
        (best_inlier < param.min_inlier || best_mean_error > param.error_threshold/param.min_inlier));

    // 4. 最终仿射重算
    if (best_inlier < 3) {
        poi.deformation = DeformationVector2D();
        poi.result.zncc = -2.f;
        poi.result.feature = 0;
        poi.result.iteration = trial_counter;
        return;
    }
    float A[9]={0}, Bx[3]={0}, By[3]={0};
    for (int k = 0; k < best_inlier; ++k) {
        int i = best_inlier_indices[k];
        float x = ref_X[i], y = ref_Y[i];
        float v[3] = {x, y, 1.f};
        for (int r = 0; r < 3; ++r) for (int c = 0; c < 3; ++c)
            A[r*3+c] += v[r]*v[c];
        for (int r = 0; r < 3; ++r) {
            Bx[r] += v[r]*tar_X[i];
            By[r] += v[r]*tar_Y[i];
        }
    }
    float invA[9];
    float det = A[0]*A[4]*A[8] + A[1]*A[5]*A[6] + A[2]*A[3]*A[7]
              - A[0]*A[5]*A[7] - A[1]*A[3]*A[8] - A[2]*A[4]*A[6];
    bool ok = fabs(det)>=1e-6f;
    float affine_x[3]={0}, affine_y[3]={0};
    if (ok) {
        invA[0]=(A[4]*A[8]-A[5]*A[7])/det; invA[1]=(A[2]*A[7]-A[1]*A[8])/det; invA[2]=(A[1]*A[5]-A[2]*A[4])/det;
        invA[3]=(A[5]*A[6]-A[3]*A[8])/det; invA[4]=(A[0]*A[8]-A[2]*A[6])/det; invA[5]=(A[2]*A[3]-A[0]*A[5])/det;
        invA[6]=(A[3]*A[7]-A[4]*A[6])/det; invA[7]=(A[1]*A[6]-A[0]*A[7])/det; invA[8]=(A[0]*A[4]-A[1]*A[3])/det;
        for (int r=0;r<3;++r) for(int j=0;j<3;++j) {
            affine_x[r]+=invA[r*3+j]*Bx[j]; affine_y[r]+=invA[r*3+j]*By[j];
        }
    }
    // 仿射参数写回（OpenCorr风格）
    poi.deformation.u   = affine_x[2];
    poi.deformation.ux  = affine_x[0] - 1.f;
    poi.deformation.uy  = affine_x[1];
    poi.deformation.v   = affine_y[2];
    poi.deformation.vx  = affine_y[0];
    poi.deformation.vy  = affine_y[1] - 1.f;
    poi.deformation.uxx = poi.deformation.uxy = poi.deformation.uyy = 0.f;
    poi.deformation.vxx = poi.deformation.vxy = poi.deformation.vyy = 0.f;

    // debug/特征统计
    float dist_sum = 0.0f, dist_max = 0.0f, dist_min = 1e10f;
    for (int i = 0; i < neighbor_num; ++i) {
        dist_sum += neighbor_dist[i];
        if (neighbor_dist[i] > dist_max) dist_max = neighbor_dist[i];
        if (neighbor_dist[i] < dist_min) dist_min = neighbor_dist[i];
    }
    float dist_mean = neighbor_num > 0 ? dist_sum / neighbor_num : 0.0f;
    poi.result.feature = best_inlier;
    poi.result.zncc = 0.f;
    poi.result.u0 = dist_mean;
    poi.result.v0 = dist_max;
    poi.result.iteration = best_trial_counter;
    poi.result.convergence = best_mean_error;
}

// --- SiftAffineBatchGpu实现 ---
SiftAffineBatchGpu::SiftAffineBatchGpu(const SiftAffineParam& param)
    : param_(param)
{
}

SiftAffineBatchGpu::~SiftAffineBatchGpu() { release_cuda(); }


// KDTree/knnSearch/暴力补齐（OpenCorr式）+分配CUDA内存
void SiftAffineBatchGpu::prepare_cuda(const SiftFeature2D* ref_kp, const SiftFeature2D* tar_kp, int num_kp, hipStream_t stream) {
    release_cuda();
    num_kp_ = num_kp;
    hipMalloc(&d_ref_kp, num_kp_ * sizeof(SiftFeature2D));
    hipMalloc(&d_tar_kp, num_kp_ * sizeof(SiftFeature2D));
    hipMemcpy(d_ref_kp, ref_kp, num_kp_*sizeof(SiftFeature2D), hipMemcpyHostToDevice);
    hipMemcpy(d_tar_kp, tar_kp, num_kp_*sizeof(SiftFeature2D), hipMemcpyHostToDevice);

    neighbor_idx_.clear();
    neighbor_dist_.clear();

    SiftKeypointCloud cloud{ref_kp, size_t(num_kp)};
    SiftKDTree kdtree(2, cloud, nanoflann::KDTreeSingleIndexAdaptorParams(10));
    kdtree.buildIndex();

    int max_neighbor = 30;
    neighbor_idx_.resize(poi_list_.size() * max_neighbor, -1);
    neighbor_dist_.resize(poi_list_.size() * max_neighbor, 1e10f);

    for (size_t pi = 0; pi < poi_list_.size(); ++pi) {
        float query_pt[2] = {poi_list_[pi].x, poi_list_[pi].y};
        std::vector<nanoflann::ResultItem<uint32_t, float>> ret_matches;

        // 1. radiusSearch
        nanoflann::SearchParameters params;
        params.sorted = false;
        kdtree.radiusSearch(query_pt, param_.kd_radius * param_.kd_radius, ret_matches, params);

        int got = 0;
        for (auto& m : ret_matches) {
            if (got >= max_neighbor) break;
            neighbor_idx_[pi*max_neighbor + got] = int(m.first);
            neighbor_dist_[pi*max_neighbor + got] = std::sqrt(m.second);
            got++;
        }

        // 2. knnSearch if not enough
        if (got < param_.min_inlier) {
            std::vector<uint32_t> knn_idx(max_neighbor);
            std::vector<float> knn_dist(max_neighbor);
            size_t found = kdtree.knnSearch(query_pt, max_neighbor, knn_idx.data(), knn_dist.data());
            for (size_t k = got; k < std::min(found, size_t(max_neighbor)); ++k) {
                neighbor_idx_[pi*max_neighbor + k] = int(knn_idx[k]);
                neighbor_dist_[pi*max_neighbor + k] = std::sqrt(knn_dist[k]);
            }
            got = int(std::max(got, int(found)));
        }

        // 3. 暴力补齐
        if (got < param_.min_inlier) {
            std::vector<std::pair<int, float>> dist_vec;
            for (int ki = 0; ki < num_kp_; ++ki) {
                float dx = ref_kp[ki].x - poi_list_[pi].x;
                float dy = ref_kp[ki].y - poi_list_[pi].y;
                float dist = std::sqrt(dx*dx + dy*dy);
                dist_vec.emplace_back(ki, dist);
            }
            std::sort(dist_vec.begin(), dist_vec.end(), [](auto& a, auto& b){ return a.second < b.second; });
            int fill = got;
            for (size_t k = 0; fill < param_.min_inlier && k < dist_vec.size() && fill < max_neighbor; ++k) {
                neighbor_idx_[pi*max_neighbor + fill] = dist_vec[k].first;
                neighbor_dist_[pi*max_neighbor + fill] = dist_vec[k].second;
                fill++;
            }
        }
        for (int k = 0; k < max_neighbor; ++k) {
            if (neighbor_idx_[pi*max_neighbor + k] < 0) {
                neighbor_dist_[pi*max_neighbor + k] = 1e10f;
            }
        }
    }

    // 分配并拷贝到device
    hipMalloc(&d_neighbor_idx_, neighbor_idx_.size() * sizeof(int));
    hipMalloc(&d_neighbor_dist_, neighbor_dist_.size() * sizeof(float));
    hipMemcpy(d_neighbor_idx_, neighbor_idx_.data(), neighbor_idx_.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_neighbor_dist_, neighbor_dist_.data(), neighbor_dist_.size()*sizeof(float), hipMemcpyHostToDevice);
    hipStreamSynchronize(stream);
}

void SiftAffineBatchGpu::compute_batch_cuda(CudaPOI2D* pois, int N, hipStream_t stream) {
    CudaPOI2D* d_pois;
    hipMalloc(&d_pois, N*sizeof(CudaPOI2D));
    hipMemcpyAsync(d_pois, pois, N*sizeof(CudaPOI2D), hipMemcpyHostToDevice, stream);

    SiftAffineRansacParam d_param;
    d_param.trial_number = param_.trial_number;
    d_param.sample_number = param_.sample_number;
    d_param.error_threshold = param_.error_threshold;
    d_param.min_inlier = param_.min_inlier;

    int max_neighbor = 30;
    int block = 128, grid = (N + block - 1) / block;
    estimate_affine_opencorr_kernel<<<grid, block, 0, stream>>>(
        d_ref_kp, d_tar_kp,
        d_neighbor_idx_, d_neighbor_dist_, max_neighbor,
        d_pois, N, d_param, 123456789ULL
    );

    hipMemcpyAsync(pois, d_pois, N*sizeof(CudaPOI2D), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    hipFree(d_pois);
}

void SiftAffineBatchGpu::release_cuda() {
    if (d_ref_kp) hipFree(d_ref_kp);
    if (d_tar_kp) hipFree(d_tar_kp);
    if (d_neighbor_idx_) hipFree(d_neighbor_idx_);
    if (d_neighbor_dist_) hipFree(d_neighbor_dist_);
    d_ref_kp = d_tar_kp = nullptr;
    d_neighbor_idx_ = nullptr;
    d_neighbor_dist_ = nullptr;
}

} // namespace StudyCorr